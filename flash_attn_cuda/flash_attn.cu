#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32
#define HEAD_DIM 64

__global__ void flash_attn_kernel(
    const float* __restrict__ query,   // [seq_len, head_dim]
    const float* __restrict__ key,     // [seq_len, head_dim]
    const float* __restrict__ value,   // [seq_len, head_dim]
    float* __restrict__ output,        // [seq_len, head_dim]
    int seq_len,
    int head_dim
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if (i >= seq_len) return;


    float q_i[HEAD_DIM];
    for (int d = 0; d < head_dim; ++d) {
        q_i[d] = query[i * head_dim + d];
    }

    float max_score = -1e9f;
    float scores[1024];  
    float exp_scores[1024];
    float sum_exp = 0.0f;


    for (int j = 0; j < seq_len; ++j) {
        float dot = 0.0f;
        for (int d = 0; d < head_dim; ++d) {
            dot += q_i[d] * key[j * head_dim + d];
        }
        scores[j] = dot;
        if (dot > max_score) max_score = dot;
    }

    for (int j = 0; j < seq_len; ++j) {
        exp_scores[j] = expf(scores[j] - max_score);
        sum_exp += exp_scores[j];
    }

    float out[HEAD_DIM] = {0.0f};

    for (int j = 0; j < seq_len; ++j) {
        float weight = exp_scores[j] / sum_exp;
        for (int d = 0; d < head_dim; ++d) {
            out[d] += weight * value[j * head_dim + d];
        }
    }

    for (int d = 0; d < head_dim; ++d) {
        output[i * head_dim + d] = out[d];
    }
}
